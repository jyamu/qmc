
#include <hip/hip_runtime.h>
#include <cstdio>
#include <unistd.h>
#include <sstream>
#include <vector>
#include <iostream>

template<typename T, int BS>
__global__ void
block_inverse (float* A, int N, int stride)
{
  __shared__ unsigned int ipiv[BS];
  __shared__ unsigned int kb;
  __shared__ T maxval[BS], mask[BS], pivotInv;
  __shared__ T Arowk[BS], Acolk[BS];
  ipiv[threadIdx.x] = threadIdx.x;
  mask[threadIdx.x] = (T)1.0;
  __syncthreads();
  unsigned int tid = threadIdx.x;
  for (int k=0; k<N; k++)
  {
    // First, find locate of maximum of kth column, excluding the
    // first k rows through the mask.
    maxval[tid] = mask[tid] * fabsf(A[tid*stride + k]);
    __syncthreads();
    for (int bs = BS>>1; bs>0; bs=bs>>1)
    {
      if (tid < bs)
        maxval[tid] =  max(maxval[tid], maxval[tid+bs]);
      __syncthreads();
    }
    if ((mask[tid] * fabsf(A[tid*stride + k])) > 0.999* maxval[0])
    {
      kb = tid;
      pivotInv = (T)1.0/A[tid*stride + k];
    }
    __syncthreads();
    // HACK HACK HACK
    //kb = k;
    //pivotInv = 1.0f/A[k*stride + k];
    //__syncthreads();
    // Now kb holds pivot row and pivot the value
    // Swap rows
    T tmp = A[k*stride+tid];
    A[k*stride +tid] = A[kb*stride+tid];
    A[kb*stride+tid] = tmp;
    // Swap pivot
    if (tid == 0)
    {
      int itmp = ipiv[kb];
      ipiv[kb] = ipiv[k];
      ipiv[k]  = itmp;
    }
    __syncthreads();
    // Col k update
    if (tid != k)
      A[stride*tid+k] = -pivotInv*A[stride*tid+k];
    else
      A[stride*k+k] = (T)0.0;
    __syncthreads();
    // Rank-1 update
    Arowk[tid] = A[stride*k   + tid];
    Acolk[tid] = A[stride*tid +   k];
    __syncthreads();
    for (int i=0; i<N; i++)
      A[i*stride+tid] += Arowk[tid]*Acolk[i];
    __syncthreads();
    // Row k update
    if (tid != k)
      A[k*stride+tid] *= pivotInv;
    else
    {
      A[k*stride+k] = pivotInv;
      mask[k] = 0.0;
    }
    __syncthreads();
  }
  // Finally, do backward pivoting
  for (int i=0; i<N; i++)
  {
    Arowk[tid] = A[i*stride+tid];
    __syncthreads();
    A[i*stride+ipiv[tid]] = Arowk[tid];
  }
}


template<typename T, int BS>
__device__ T
block_inverse2 (T A[BS][BS+1])
{
  __shared__ unsigned int ipiv[BS];
  __shared__ unsigned int kb;
  __shared__ T maxval[BS], mask[BS], pivotInv;
  __shared__ T Arowk[BS], Acolk[BS];
  bool write = threadIdx.y == 0;
  ipiv[threadIdx.x] = threadIdx.x;
  mask[threadIdx.x] = (T)1.0;
  __syncthreads();
  unsigned int tid = threadIdx.x;
  __shared__ T det;
  if (tid == 0)
    det = (T)1.0;
  for (int k=0; k<BS; k++)
  {
    // First, find locate of maximum of kth column, excluding the
    // first k rows through the mask.
    if (write)
      maxval[tid] = mask[tid] * fabsf(A[tid][k]);
    __syncthreads();
    for (int bs = BS>>1; bs>0; bs=bs>>1)
    {
      if (tid < bs && write)
        maxval[tid] =  max(maxval[tid], maxval[tid+bs]);
      __syncthreads();
    }
    if ((mask[tid] * fabsf(A[tid][k])) == maxval[0] && write)
    {
      kb = tid;
      pivotInv = (T)1.0/A[tid][k];
      if (kb == k)	det *= A[tid][k];
      else              det *= -A[tid][k];
    }
    __syncthreads();
    // Now kb holds pivot row and pivot the value
    // Swap rows
    if (write)
    {
      T tmp = A[k][tid];
      A[k][tid] = A[kb][tid];
      A[kb][tid] = tmp;
    }
    // Swap pivot
    if (tid == 0 && write)
    {
      int itmp = ipiv[kb];
      ipiv[kb] = ipiv[k];
      ipiv[k]  = itmp;
    }
    __syncthreads();
    // Col k update
    if (write)
    {
      if (tid != k)
        A[tid][k] = -pivotInv*A[tid][k];
      else
        A[k][k] = (T)0.0;
    }
    __syncthreads();
    // Rank-1 update
    Arowk[tid] = A[k][tid];
    Acolk[tid] = A[tid][k];
    __syncthreads();
    for (int i=0; i<BS; i+=blockDim.y)
      A[i+threadIdx.y][tid] += Arowk[tid]*Acolk[i+threadIdx.y];
    __syncthreads();
    // Row k update
    if (write)
    {
      if (tid != k)
        A[k][tid] *= pivotInv;
      else
      {
        A[k][k] = pivotInv;
        mask[k] = 0.0;
      }
    }
    __syncthreads();
  }
  // Finally, do backward pivoting
  for (int i=0; i<BS; i++)
  {
    if (write)
      Arowk[tid] = A[i][tid];
    __syncthreads();
    if (write)
      A[i][ipiv[tid]] = Arowk[tid];
  }
  return det;
}


template<typename T, int BS>
__device__ void block_mul2 (T A[BS][BS+1],
                            T B[BS][BS+1],
                            T C[BS][BS+1])
{
  int tid = threadIdx.x;
  for (int row=0; row<BS; row++)
    C[row][tid] = (T)0.0;
  __syncthreads();
  for (int k=0; k<BS; k++)
    for (int i=0; i<BS; i++)
      C[i][tid] += A[i][k]*B[k][tid];
}


template<typename T, int BS>
__device__ void block_mul_add2 (T A[BS][BS+1],
                                T B[BS][BS+1],
                                T *C, int Cstride)
{
  int tid = threadIdx.x;
  __shared__ T Crow[2][BS];
  for (int i=0; i<BS; i+=blockDim.y)
  {
    Crow[threadIdx.y][tid] = C[(i+threadIdx.y)*Cstride + tid];
    for (int k=0; k<BS; k++)
      Crow[threadIdx.y][tid] += A[i+threadIdx.y][k]*B[k][tid];
    C[(i+threadIdx.y)*Cstride + tid] = Crow[threadIdx.y][tid];
  }
}

template<typename T, int BS>
__device__ void block_mul_set2 (T A[BS][BS+1],
                                T B[BS][BS+1],
                                T *C, int Cstride)
{
  int tid = threadIdx.x;
  __shared__ T Crow[2][BS];
  for (int i=0; i<BS; i+=blockDim.y)
  {
    Crow[threadIdx.y][tid] = (T)0.0;
    for (int k=0; k<BS; k++)
      Crow[threadIdx.y][tid] += A[i+threadIdx.y][k]*B[k][tid];
    C[(i+threadIdx.y)*Cstride + tid] = Crow[threadIdx.y][tid];
  }
}



template<typename T, int BS>
__device__ T
block_inverse1 (T A[BS][BS+1])
{
  __shared__ unsigned int ipiv[BS];
  __shared__ unsigned int kb;
  __shared__ T maxval[BS], mask[BS], pivotInv;
  __shared__ T Arowk[BS], Acolk[BS];
  ipiv[threadIdx.x] = threadIdx.x;
  mask[threadIdx.x] = (T)1.0;
  __syncthreads();
  unsigned int tid = threadIdx.x;
  __shared__ T det;
  if (tid == 0)
    det = (T)1.0;
  for (int k=0; k<BS; k++)
  {
    // First, find locate of maximum of kth column, excluding the
    // first k rows through the mask.
    maxval[tid] = mask[tid] * fabsf(A[tid][k]);
    __syncthreads();
    for (int bs = BS>>1; bs>0; bs=bs>>1)
    {
      if (tid < bs)
        maxval[tid] =  max(maxval[tid], maxval[tid+bs]);
      __syncthreads();
    }
    if ((mask[tid] * fabsf(A[tid][k])) == maxval[0])
    {
      kb = tid;
      pivotInv = (T)1.0/A[tid][k];
      if (kb == k)	det *= A[tid][k];
      else              det *= -A[tid][k];
    }
    __syncthreads();
    // Now kb holds pivot row and pivot the value
    // Swap rows
    T tmp = A[k][tid];
    A[k][tid] = A[kb][tid];
    A[kb][tid] = tmp;
    // Swap pivot
    if (tid == 0)
    {
      int itmp = ipiv[kb];
      ipiv[kb] = ipiv[k];
      ipiv[k]  = itmp;
    }
    __syncthreads();
    // Col k update
    if (tid != k)
      A[tid][k] = -pivotInv*A[tid][k];
    else
      A[k][k] = (T)0.0;
    __syncthreads();
    // Rank-1 update
    Arowk[tid] = A[k][tid];
    Acolk[tid] = A[tid][k];
    __syncthreads();
    for (int i=0; i<BS; i++)
      A[i][tid] += Arowk[tid]*Acolk[i];
    __syncthreads();
    // Row k update
    if (tid != k)
      A[k][tid] *= pivotInv;
    else
    {
      A[k][k] = pivotInv;
      mask[k] = 0.0;
    }
    __syncthreads();
  }
  // Finally, do backward pivoting
  for (int i=0; i<BS; i++)
  {
    Arowk[tid] = A[i][tid];
    __syncthreads();
    A[i][ipiv[tid]] = Arowk[tid];
  }
  return det;
}


template<typename T, int BS>
__device__ void block_mul (T A[BS][BS+1],
                           T B[BS][BS+1],
                           T C[BS][BS+1])
{
  int tid = threadIdx.x;
  for (int row=0; row<BS; row++)
    C[row][tid] = (T)0.0;
  __syncthreads();
  for (int k=0; k<BS; k++)
    for (int i=0; i<BS; i++)
      C[i][tid] += A[i][k]*B[k][tid];
}


template<typename T, int BS>
__device__ void block_mul_add (T A[BS][BS+1],
                               T B[BS][BS+1],
                               T *C, int Cstride)
{
  int tid = threadIdx.x;
  __shared__ T Crow[BS];
  for (int i=0; i<BS; i++)
  {
    Crow[tid] = C[i*Cstride + tid];
    for (int k=0; k<BS; k++)
      Crow[tid] += A[i][k]*B[k][tid];
    C[i*Cstride + tid] = Crow[tid];
  }
}

template<typename T, int BS>
__device__ void block_mul_set (T A[BS][BS+1],
                               T B[BS][BS+1],
                               T *C, int Cstride)
{
  int tid = threadIdx.x;
  __shared__ T Crow[BS];
  for (int i=0; i<BS; i++)
  {
    Crow[tid] = (T)0.0;
    for (int k=0; k<BS; k++)
      Crow[tid] += A[i][k]*B[k][tid];
    C[i*Cstride + tid] = Crow[tid];
  }
}




template<typename T, int BS>
__global__ void
inverse (T* A, T* work, int N, int stride)
{
  T *Atmp = work;
  T *pivot_tmp = work+N*stride;
  __shared__ T pivot[BS][BS+1], in[BS][BS+1];
  int NB = N/BS;
  if (N%BS) NB++;
  int tid = threadIdx.x;
  for (int kb=0; kb<NB; kb++)
  {
    // load pivot block
    int row = kb*BS;
    for (int j=0; j<BS; j++)
      if (row+tid < N)
        pivot[j][tid] = A[(row+j)*stride + row+tid];
    // invert pivot
    block_inverse1<T,BS> (pivot);
    // Column scaling
    int col = kb*BS;
    for (int jb=0; jb < NB; jb++)
    {
      int row = jb*BS;
      if (kb != jb)
      {
        for (int j=0; j<BS; j++)
          in[j][tid] = -A[(row+j)*stride + col + tid];
        block_mul_set<T,BS>(in, pivot, A+row*stride+col, stride);
      }
      else
      {
        for (int j=0; j<BS; j++)
          A[(row+j)*stride + col+tid] = (T)0.0;
      }
    }
    // Save pivot to global memory here!
    // We use it for temporary space in the rank-1 update
    for (int j=0; j<BS; j++)
      pivot_tmp[j*BS+tid] = pivot[j][tid];
    // Copy Ato Atmp
    for (int ib=0; ib<NB; ib++)
      for (int row=0; row<N; row++)
        Atmp[row*stride+ib*BS+tid] =  A[row*stride+ib*BS+tid];
    // Rank-1 update
    for (int ib=0; ib < NB; ib++)
    {
      for (int i=0; i<BS; i++)
        in[i][tid] = A[(ib*BS+i)*stride + kb*BS + tid];
      for (int jb=0; jb<NB; jb++)
      {
        for (int i=0; i<BS; i++)
        {
          pivot[i][tid] = A[(kb*BS+i)*stride + jb*BS + tid];
          // Atmp[(ib*BS+i)*stride + (jb*BS+tid)] =
          //   A[(ib*BS+i)*stride + (jb*BS+tid)];
        }
        block_mul_add<T,BS>(in, pivot,  Atmp+(ib*BS)*stride + jb*BS,
                            stride);
      }
    }
    // Copy Atmp back to A
    for (int ib=0; ib<NB; ib++)
      for (int row=0; row<N; row++)
        A[row*stride+ib*BS+tid] =  Atmp[row*stride+ib*BS+tid];
    // Restore pivot from global memory here!
    for (int j=0; j<BS; j++)
      pivot[j][tid] = pivot_tmp[j*BS+tid];
    // Row-scaling
    for (int ib=0; ib<NB; ib++)
    {
      int row = kb*BS;
      int col = ib*BS;
      if (kb != ib)
      {
        for (int j=0; j<BS; j++)
          in[j][tid] = A[(row+j)*stride + col+tid];
        block_mul_set<T,BS>(pivot, in, A+row*stride+col, stride);
      }
      else
      {
        for (int j=0; j<BS; j++)
          A[(row+j)*stride + col+tid] = pivot[j][tid];
      }
    }
  }
}



template<typename T, int BS>
__global__ void
inverse_many (T **A_list, T **work_list, int N, int stride)
{
  int tid = threadIdx.x;
  __shared__ T *A, *work;
  if (tid == 0 && threadIdx.y == 0)
  {
    A    = A_list[blockIdx.x];
    work = work_list[blockIdx.x];
  }
  __syncthreads();
  T *Atmp = work;
  T *pivot_tmp = work+N*stride;
  __shared__ T pivot[BS][BS+1], in[BS][BS+1];
  int NB = N/BS;
  if (N%BS) NB++;
  for (int kb=0; kb<NB; kb++)
  {
    // load pivot block
    int row = kb*BS;
    for (int j=0; j<BS; j++)
      if (row+tid < N && threadIdx.y == 0)
        pivot[j][tid] = A[(row+j)*stride + row+tid];
    // invert pivot
    block_inverse2<T,BS> (pivot);
    // Column scaling
    int col = kb*BS;
    for (int jb=0; jb < NB; jb++)
    {
      int row = jb*BS;
      if (kb != jb)
      {
        if (threadIdx.y == 0)
          for (int j=0; j<BS; j++)
            in[j][tid] = -A[(row+j)*stride + col + tid];
        __syncthreads();
        block_mul_set2<T,BS>(in, pivot, A+row*stride+col, stride);
      }
      else if (threadIdx.y == 0)
        for (int j=0; j<BS; j++)
          A[(row+j)*stride + col+tid] = (T)0.0;
    }
    // Save pivot to global memory here!
    // We use it for temporary space in the rank-1 update
    if (threadIdx.y == 0)
    {
      for (int j=0; j<BS; j++)
        pivot_tmp[j*BS+tid] = pivot[j][tid];
      // Copy Ato Atmp
      for (int ib=0; ib<NB; ib++)
        for (int row=0; row<N; row++)
          Atmp[row*stride+ib*BS+tid] =  A[row*stride+ib*BS+tid];
    }
    __syncthreads();
    // Rank-1 update
    for (int ib=0; ib < NB; ib++)
    {
      if (threadIdx.y == 0)
        for (int i=0; i<BS; i++)
          in[i][tid] = A[(ib*BS+i)*stride + kb*BS + tid];
      for (int jb=0; jb<NB; jb++)
      {
        if (threadIdx.y == 0)
          for (int i=0; i<BS; i++)
            pivot[i][tid] = A[(kb*BS+i)*stride + jb*BS + tid];
        __syncthreads();
        block_mul_add2<T,BS>(in, pivot,  Atmp+(ib*BS)*stride + jb*BS,
                             stride);
        __syncthreads();
      }
    }
    // Copy Atmp back to A
    if (threadIdx.y == 0)
      for (int ib=0; ib<NB; ib++)
        for (int row=0; row<N; row++)
          A[row*stride+ib*BS+tid] =  Atmp[row*stride+ib*BS+tid];
    // Restore pivot from global memory here!
    if (threadIdx.y == 0)
      for (int j=0; j<BS; j++)
        pivot[j][tid] = pivot_tmp[j*BS+tid];
    // Row-scaling
    for (int ib=0; ib<NB; ib++)
    {
      int row = kb*BS;
      int col = ib*BS;
      if (kb != ib)
      {
        if (threadIdx.y == 0)
          for (int j=0; j<BS; j++)
            in[j][tid] = A[(row+j)*stride + col+tid];
        __syncthreads();
        block_mul_set2<T,BS>(pivot, in, A+row*stride+col, stride);
      }
      else
      {
        if (threadIdx.y == 0)
          for (int j=0; j<BS; j++)
            A[(row+j)*stride + col+tid] = pivot[j][tid];
      }
    }
  }
}

#define MAX_BLOCKS 32

template<typename T, int BS>
__global__ void
inverse_many_pivot (T **A_list, T **work_list, int N, int stride)
{
  int tid = threadIdx.x;
  __shared__ T *A, *work;
  T maxdet, blockdet, det;
  __shared__ int ipiv[MAX_BLOCKS];
  if (tid == 0)
  {
    A    = A_list[blockIdx.x];
    work = work_list[blockIdx.x];
    det = (T)1.0;
  }
  ipiv[tid] = tid;
  ipiv[tid+BS] = tid+BS;
  __syncthreads();
  T *Atmp = work;
  T *pivot_tmp = work+N*stride;
  __shared__ T pivot[BS][BS+1], in[MAX_BLOCKS][BS+1];
  int NB = N/BS;
  if (N%BS) NB++;
  for (int kb=0; kb<NB; kb++)
  {
    int imax = kb;
    maxdet = (T)0.0;
    // Find pivot block
    for (int block=kb; block<NB; block++)
    {
      // load pivot block
      int row = block*BS;
      for (int j=0; j<BS; j++)
        if (row+tid < N)
          in[j][tid] = A[(row+j)*stride + kb*BS + tid];
      __syncthreads();
      // invert pivot
      blockdet = block_inverse1<T,BS> (in);
      __syncthreads();
      if (fabs(blockdet) > fabs(maxdet))
      {
        //if (block == kb) {
        imax = block;
        maxdet = blockdet;
        for (int j=0; j<BS; j++)
          pivot[j][tid] = in[j][tid];
      }
    }
    // Now, swap row blocks
    for (int j=0; j<BS; j++)
    {
      int rowa = kb   * BS + j;
      int rowb = imax * BS + j;
      for (int n=0; n<NB; n++)
      {
        int col = n*BS + tid;
        T tmp = A[rowa*stride + col];
        __syncthreads();
        A[rowa*stride + col] = A[rowb*stride + col];
        __syncthreads();
        A[rowb*stride + col] = tmp;
      }
    }
    if (tid == 0)
    {
      int tmp = ipiv[kb];
      ipiv[kb] = ipiv[imax];
      ipiv[imax] = tmp;
      if (imax == kb)
        det *= blockdet;
      else
        det *= -blockdet;
    }
    // Column scaling
    int col = kb*BS;
    for (int jb=0; jb < NB; jb++)
    {
      int row = jb*BS;
      if (kb != jb)
      {
        for (int j=0; j<BS; j++)
          in[j][tid] = -A[(row+j)*stride + col + tid];
        block_mul_set<T,BS>(in, pivot, A+row*stride+col, stride);
      }
      else
      {
        for (int j=0; j<BS; j++)
          A[(row+j)*stride + col+tid] = (T)0.0;
      }
    }
    // Save pivot to global memory here!
    // We use it for temporary space in the rank-1 update
    for (int j=0; j<BS; j++)
      pivot_tmp[j*BS+tid] = pivot[j][tid];
    // Copy Ato Atmp
    for (int ib=0; ib<NB; ib++)
      for (int row=0; row<N; row++)
        Atmp[row*stride+ib*BS+tid] =  A[row*stride+ib*BS+tid];
    // Rank-1 update
    for (int ib=0; ib < NB; ib++)
    {
      for (int i=0; i<BS; i++)
        in[i][tid] = A[(ib*BS+i)*stride + kb*BS + tid];
      for (int jb=0; jb<NB; jb++)
      {
        for (int i=0; i<BS; i++)
          pivot[i][tid] = A[(kb*BS+i)*stride + jb*BS + tid];
        block_mul_add<T,BS>(in, pivot,  Atmp+(ib*BS)*stride + jb*BS,
                            stride);
      }
    }
    // Copy Atmp back to A
    for (int ib=0; ib<NB; ib++)
      for (int row=0; row<N; row++)
        A[row*stride+ib*BS+tid] =  Atmp[row*stride+ib*BS+tid];
    // Restore pivot from global memory here!
    for (int j=0; j<BS; j++)
      pivot[j][tid] = pivot_tmp[j*BS+tid];
    // Row-scaling
    for (int ib=0; ib<NB; ib++)
    {
      int row = kb*BS;
      int col = ib*BS;
      if (kb != ib)
      {
        for (int j=0; j<BS; j++)
          in[j][tid] = A[(row+j)*stride + col+tid];
        block_mul_set<T,BS>(pivot, in, A+row*stride+col, stride);
      }
      else
      {
        for (int j=0; j<BS; j++)
          A[(row+j)*stride + col+tid] = pivot[j][tid];
      }
    }
  }
  // Note:  the following assumes that N <= BS*BS
  // Finally, reverse pivoting
  for (int row=0; row<N; row++)
  {
    for (int block=0; block < NB; block++)
      in[block][tid] = A[row*stride + BS*block + tid];
    __syncthreads();
    for (int block=0; block<NB; block++)
      A[row*stride + BS*ipiv[block] + tid] = in[block][tid];
  }
}


template<typename T, int BS>
__global__ void
inverse_many_naive (T **A_list, T **work_list, int N, int stride)
{
  int tid = threadIdx.x;
  __shared__ T *A;
  if (tid == 0)
    A    = A_list[blockIdx.x];
  __syncthreads();
  __shared__ T colk[BS], rowk[BS];
  for (int k=0; k<N; k++)
  {
    rowk[tid] = A[k*stride+tid];
    __syncthreads();
    T pivInv = 1.0/rowk[k];
    __syncthreads();
    if (tid == k)
    {
      rowk[k] = T(0);
    }
    // Column scaling
    colk[tid] = (tid==k) ? T(0) : -pivInv*A[tid*stride+k];
    __syncthreads();
    A[tid*stride+k] = colk[tid];
    __syncthreads();
    // Rank-1 update
    for (int j=0; j<N; j++)
      A[j*stride+tid] += colk[j]*rowk[tid];
    __syncthreads();
    // Row scaling
    A[k*stride + tid] = pivInv * ((tid==k) ? 1.0 : rowk[tid]);
    __syncthreads();
  }
}




template<typename T, int BS>
__global__ void
inverse_many_naive_pivot (T **A_list, T **work_list, int N, int stride)
{
  int tid = threadIdx.x;
  __shared__ T *A;
  if (tid == 0)
    A    = A_list[blockIdx.x];
  __syncthreads();
  __shared__ int kbar, ipiv[BS];
  __shared__ T colk[BS], rowk[BS];
  __shared__ short imax[BS];
  ipiv[tid] = tid;
  __syncthreads();
  for (int k=0; k<N; k++)
  {
    // Find location of largest element in the column at or below k
    colk[tid] = (tid < k) ? 0.0 : fabs(A[tid*stride+k]);
    rowk[tid] = colk[tid];
    __syncthreads();
    int skip = 1<<((int)ceil(log2((double)BS)-1.0e-6)-1);
    imax[tid] = tid;
    __syncthreads();
    for (; skip>0; skip>>=1)
    {
      if (tid < skip && (tid+skip)<N)
        // colk[tid] = max(colk[tid],colk[tid+skip]);
        if (colk[tid+skip] > colk[tid])
        {
          imax[tid] = imax[tid+skip];
          colk[tid] = colk[tid+skip];
        }
      __syncthreads();
    }
    if (tid == 0)
    {
      kbar = imax[0];
      int i = ipiv[kbar];
      ipiv[kbar] = ipiv[k];
      ipiv[k] = i;
    }
    // if (rowk[tid] == colk[0]) {
    //   kbar = tid;
    //   int i = ipiv[tid];
    //   ipiv[tid] = ipiv[k];
    //   ipiv[k]   = i;
    // }
    __syncthreads();
    // Swap rows
    rowk[tid] = A[kbar*stride + tid];
    colk[tid] = A[k   *stride + tid];
    __syncthreads();
    A[k   *stride + tid] = rowk[tid];
    A[kbar*stride + tid] = colk[tid];
    __syncthreads();
    T pivInv = 1.0/rowk[k];
    __syncthreads();
    if (tid == k)
    {
      rowk[k] = T(0);
    }
    // Column scaling
    colk[tid] = (tid==k) ? T(0) : -pivInv*A[tid*stride+k];
    __syncthreads();
    A[tid*stride+k] = colk[tid];
    __syncthreads();
    // Rank-1 update
    for (int j=0; j<N; j++)
      A[j*stride+tid] += colk[j]*rowk[tid];
    __syncthreads();
    // Row scaling
    A[k*stride + tid] = pivInv * ((tid==k) ? 1.0 : rowk[tid]);
    __syncthreads();
  }
  // Now, permute columns one row at a time in shared memory
  for (int k=0; k<N; k++)
  {
    rowk[tid] = A[k*stride+tid];
    __syncthreads();
    colk[ipiv[tid]] = rowk[tid];
    __syncthreads();
    A[k*stride+tid] = colk[tid];
    __syncthreads();
  }
}



template<typename T, int BS>
__global__ void
complex_inverse_many_naive_pivot (T **A_list, T **work_list, int N, int stride)
{
  int tid = threadIdx.x;
  __shared__ T *A;
  unsigned str = 2*stride;
  if (tid == 0)
    A    = A_list[blockIdx.x];
  __syncthreads();
  __shared__ int kbar, ipiv[BS];
  __shared__ T colk[2*BS], rowk[2*BS];
  __shared__ short imax[BS];
  ipiv[tid] = tid;
  __syncthreads();
  for (int k=0; k<N; k++)
  {
    // Find location of largest element in the column at or below k
    T re = A[tid*str+2*k];
    T im = A[tid*str+2*k+1];
    colk[tid] = (tid < k) ? 0.0 : re*re + im*im;
    __syncthreads();
    int skip = 1<<((int)ceil(log2((double)BS)-1.0e-6)-1);
    imax[tid] = tid;
    __syncthreads();
    for (; skip>0; skip>>=1)
    {
      if (tid < skip && (tid+skip)<N)
        // colk[tid] = max(colk[tid],colk[tid+skip]);
        if (colk[tid+skip] > colk[tid])
        {
          imax[tid] = imax[tid+skip];
          colk[tid] = colk[tid+skip];
        }
      __syncthreads();
    }
    if (tid == 0)
    {
      kbar = imax[0];
      int i = ipiv[kbar];
      ipiv[kbar] = ipiv[k];
      ipiv[k] = i;
    }
    __syncthreads();
    // Swap rows
    rowk[tid   ] = A[kbar*str + tid];
    rowk[tid+BS] = A[kbar*str + tid + BS];
    colk[tid   ] = A[k   *str + tid];
    colk[tid+BS] = A[k   *str + tid + BS];
    __syncthreads();
    A[k   *str + tid   ] = rowk[tid];
    A[k   *str + tid+BS] = rowk[tid+BS];
    A[kbar*str + tid   ] = colk[tid];
    A[kbar*str + tid+BS] = colk[tid+BS];
    __syncthreads();
    re = rowk[2*k+0];
    im = rowk[2*k+1];
    T nrmInv = 1.0/(re*re + im*im);
    T pivInv_re =  nrmInv * re;
    T pivInv_im = -nrmInv * im;
    __syncthreads();
    if (tid == k)
    {
      rowk[2*k+0] = T(0);
      rowk[2*k+1] = T(0);
    }
    // Column scaling
    re = A[tid*str+2*k+0];
    im = A[tid*str+2*k+1];
    colk[2*tid+0] = -(pivInv_re*re - pivInv_im*im);
    colk[2*tid+1] = -(pivInv_im*re + pivInv_im*re);
    if (tid == k)
    {
      colk[2*tid+0] = colk[2*tid+1] = T(0);
    }
    __syncthreads();
    A[tid*str+k   ] = colk[tid];
    A[tid*str+k+BS] = colk[tid+BS];
    __syncthreads();
    // Rank-1 update
    for (int j=0; j<N; j++)
    {
      re = colk[2*j+0]*rowk[2*tid+0] - colk[2*j+1]*rowk[2*tid+1];
      im = colk[2*j+0]*rowk[2*tid+1] + colk[2*j+1]*rowk[2*tid+0];
      A[j*str+2*tid  ] += re;
      A[j*str+2*tid+1] += im;
    }
    __syncthreads();
    // Row scaling
    re = pivInv_re * rowk[2*tid+0] - pivInv_im * rowk[2*tid+1];
    im = pivInv_re * rowk[2*tid+1] + pivInv_im * rowk[2*tid+0];
    if (tid == k)
    {
      re = im = T(0);
    }
    A[k*str + 2*tid+0] = re;
    A[k*str + 2*tid+1] = im;
    __syncthreads();
  }
  // Now, permute columns one row at a time in shared memory
  for (int k=0; k<N; k++)
  {
    rowk[tid   ] = A[k*str+tid   ];
    rowk[tid+BS] = A[k*str+tid+BS];
    __syncthreads();
    colk[2*ipiv[tid]+0] = rowk[2*tid+0];
    colk[2*ipiv[tid]+1] = rowk[2*tid+1];
    __syncthreads();
    A[k*str+tid   ] = colk[tid   ];
    A[k*str+tid+BS] = colk[tid+BS];
    __syncthreads();
  }
}







#define CONVERT_BS 256


template<typename Tdest, typename Tsrc>
__global__ void
convert (Tdest **dest_list, Tsrc **src_list, int len)
{
  __shared__ Tsrc *mysrc;
  __shared__ Tdest *mydest;
  if (threadIdx.x ==0)
  {
    mysrc = src_list[blockIdx.y];
    mydest = dest_list[blockIdx.y];
  }
  __syncthreads();
  int i = blockIdx.x * CONVERT_BS + threadIdx.x;
  if (i < len)
    mydest[i] = (Tdest)mysrc[i];
}


template<typename Tdest, typename Tsrc>
__global__ void
convert (Tdest **dest_list, Tsrc **src_list,
         int dest_rows, int dest_cols, int dest_rowstride,
         int src_rows,  int src_cols,  int src_rowstride)
{
  __shared__ Tsrc *mysrc;
  __shared__ Tdest *mydest;
  if (threadIdx.x ==0)
  {
    mysrc = src_list[blockIdx.y];
    mydest = dest_list[blockIdx.y];
  }
  __syncthreads();
  int i = blockIdx.x * CONVERT_BS + threadIdx.x;
  int row = i / dest_rowstride;
  int col = i - row*dest_rowstride;
  if (row < dest_rows && col < dest_rowstride)
  {
    if (col < src_cols && row < src_rows)
      mydest[i] = (Tdest)mysrc[row*src_rowstride + col];
    else
      mydest[i] = (row == col) ? (Tdest)1.0 : (Tdest)0.0;
  }
}





#define INVERSE_BS 16

void
cuda_inverse_many (float **Alist_d, float **worklist_d,
                   int N, int num_mats)
{
  dim3 dimBlock(INVERSE_BS,2);
  dim3 dimGrid(num_mats);
  inverse_many_pivot<float,INVERSE_BS><<<dimGrid,dimBlock>>>
  (Alist_d, worklist_d, N, N);
}


size_t
cuda_inverse_many_worksize(int N)
{
  return (N * N + INVERSE_BS*INVERSE_BS);
}


size_t
cuda_inverse_many_double_worksize(int N)
{
  int N_double = ((N+INVERSE_BS-1)/INVERSE_BS) * INVERSE_BS;
  return 2*(2*N_double*N_double + INVERSE_BS*INVERSE_BS);
}

void
cuda_inverse_many_double (float *Alist_d[], float *worklist_d[],
                          int N, int num_mats)
{
  dim3 dimBlockConvert (CONVERT_BS);
  dim3 dimGridConvert(N*N/CONVERT_BS, num_mats);
  if (N*N % CONVERT_BS)
    dimGridConvert.x++;
  convert<<<dimGridConvert,dimBlockConvert>>>
  ((double**)worklist_d, Alist_d, N*N);
  float *Alist_new[num_mats], *Alist_h[num_mats];
  float *worklist_h[num_mats];
  double *worklist_double_h[num_mats];
  hipMemcpy (worklist_h, worklist_d, num_mats*sizeof(float*),
              hipMemcpyDeviceToHost);
  hipMemcpy (Alist_h, Alist_d, num_mats*sizeof(float*),
              hipMemcpyDeviceToHost);
  for (int i=0; i<num_mats; i++)
  {
    Alist_new[i] = worklist_h[i];
    worklist_double_h[i] = (double*)(worklist_h[i]) +N*N;
  }
  hipMemcpyAsync (worklist_d, worklist_double_h, num_mats*sizeof(double*),
                   hipMemcpyHostToDevice);
  hipMemcpyAsync (Alist_d, Alist_new, num_mats*sizeof(double*),
                   hipMemcpyHostToDevice);
  dim3 dimBlock(INVERSE_BS,2);
  dim3 dimGrid(num_mats);
  inverse_many_pivot<double,INVERSE_BS><<<dimGrid,dimBlock>>>
  ((double**)Alist_d, (double**)worklist_d, N, N);
  hipMemcpyAsync (Alist_d, Alist_h, num_mats*sizeof(float*),
                   hipMemcpyHostToDevice);
  hipMemcpyAsync (worklist_d, worklist_h, num_mats*sizeof(float*),
                   hipMemcpyHostToDevice);
  convert<<<dimGridConvert,dimBlockConvert>>>
  (Alist_d, (double**) worklist_d, N*N);
}



// VERY slow implementation used only for debugging.
template<typename T1, typename T2, int BS>
__global__ void
check_inv (T1 **A, T2 **B, int N, int Astride, int Bstride)
{
  int tid = threadIdx.x;
  __shared__ T1 *myA;
  __shared__ T2 *myB;
  if (tid == 0)
  {
    myA = A[blockIdx.x];
    myB = B[blockIdx.x];
  }
  __syncthreads();
  __shared__ double AB[BS];
  int NB = (N+BS-1)/BS;
  bool error = false;
  for (int row=0; row<N; row++)
    for (int col=0; col<N; col++)
    {
      AB[tid] = 0.0;
      int off = tid;
      for (int block=0; block<NB; block++)
      {
        if (off < N)
          AB[tid] += myA[row*Astride+off]*myB[off*Bstride+col];
        off += BS;
      }
      // Now do reduction
      int skip = 1<<((int)ceil(log2((double)BS)-1.0e-6)-1);
      for (; skip>0; skip>>=1)
      {
        if (tid < skip)
          AB[tid] += AB[tid+skip];
        __syncthreads();
      }
      double expected = (row==col) ? 1.0 : 0.0;
      error = error || (fabs(AB[tid]-expected) > 1.0e-6);
    }
  if (tid == 0)
    A[blockIdx.x] = (T1*)(error ? 1 : 0);
  __syncthreads();
}




void
cuda_inverse_many_double (float *Alist_d[], float *worklist_d[],
                          int N, int row_stride, int num_mats)
{
  int N_double = ((N + INVERSE_BS-1)/INVERSE_BS)*INVERSE_BS;
  dim3 dimBlockConvert (CONVERT_BS);
  dim3 dimGridConvert((N_double*N_double+(CONVERT_BS-1))/CONVERT_BS,
                      num_mats);
  convert<<<dimGridConvert,dimBlockConvert>>>
  ((double**)worklist_d, Alist_d,
   N_double, N_double, N_double,
   N, N, row_stride);
  // We need to replace the pointers to the single-precision A
  // with the double-precision version we just converted.  We
  // Also need generate a new set of workspace pointers to point
  // to the region after the double-precision A.
  float *Alist_new[num_mats], *Alist_h[num_mats];
  float *worklist_h[num_mats];
  double *worklist_double_h[num_mats];
  float *bad_inverse[num_mats];
  // Save the original pointer lists on the host
  hipMemcpy (worklist_h, worklist_d, num_mats*sizeof(float*),
              hipMemcpyDeviceToHost);
  hipMemcpy (Alist_h, Alist_d, num_mats*sizeof(float*),
              hipMemcpyDeviceToHost);
  // Create new pointers as discussed above
  for (int i=0; i<num_mats; i++)
  {
    Alist_new[i] = worklist_h[i];
    worklist_double_h[i] = (double*)(worklist_h[i]) +N_double*N_double;
  }
  hipMemcpyAsync (worklist_d, worklist_double_h, num_mats*sizeof(double*),
                   hipMemcpyHostToDevice);
  hipMemcpyAsync (Alist_d, Alist_new, num_mats*sizeof(double*),
                   hipMemcpyHostToDevice);
  // Do the inversion in double-precision
  dim3 dimGrid(num_mats);
  // This appears to cause NANs for certain matrix sizes on occasion.
  // Check the pivoting algorithm.
//   dim3 dimBlock(INVERSE_BS,2);
//   inverse_many<double,INVERSE_BS><<<dimGrid,dimBlock>>>
//     ((double**)Alist_d, (double**)worklist_d, N_double, N_double);
//   dim3 dimBlock(INVERSE_BS);
//   inverse_many_pivot<double,INVERSE_BS><<<dimGrid,dimBlock>>>
//     ((double**)Alist_d, (double**)worklist_d, N_double, N_double);
  int NB = (N+15)/16;
  int BS=0;
  dim3 dimBlock(NB*16);
  switch (NB)
  {
  case 1:
    inverse_many_naive_pivot<double,16><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=16;
    break;
  case 2:
    inverse_many_naive_pivot<double,32><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=32;
    break;
  case 3:
    inverse_many_naive_pivot<double,48><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=48;
    break;
  case 4:
    inverse_many_naive_pivot<double,64><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=64;
    break;
  case 5:
    inverse_many_naive_pivot<double,80><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=80;
    break;
  case 6:
    inverse_many_naive_pivot<double,96><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=96;
    break;
  case 7:
    inverse_many_naive_pivot<double,112><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=112;
    break;
  case 8:
    inverse_many_naive_pivot<double,128><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=128;
    break;
  case 9:
    inverse_many_naive_pivot<double,144><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=144;
    break;
  case 10:
    inverse_many_naive_pivot<double,160><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=160;
    break;
  case 11:
    inverse_many_naive_pivot<double,176><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=176;
    break;
  case 12:
    inverse_many_naive_pivot<double,192><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=192;
    break;
  case 13:
    inverse_many_naive_pivot<double,208><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 208;
    break;
  case 14:
    inverse_many_naive_pivot<double,224><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=224;
    break;
  case 15:
    inverse_many_naive_pivot<double,240><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=240;
    break;
  case 16:
    inverse_many_naive_pivot<double,256><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=256;
    break;
  case 17:
    inverse_many_naive_pivot<double,272><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=272;
    break;
  case 18:
    inverse_many_naive_pivot<double,288><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=288;
    break;
  case 19:
    inverse_many_naive_pivot<double,304><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=304;
    break;
  case 20:
    inverse_many_naive_pivot<double,320><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=320;
    break;
  case 21:
    inverse_many_naive_pivot<double,336><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=336;
    break;
  case 22:
    inverse_many_naive_pivot<double,352><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 352;
    break;
  case 23:
    inverse_many_naive_pivot<double,368><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 368;
    break;
  case 24:
    inverse_many_naive_pivot<double,384><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 384;
    break;
  case 25:
    inverse_many_naive_pivot<double,400><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 400;
    break;
  case 26:
    inverse_many_naive_pivot<double,416><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 416;
    break;
  case 27:
    inverse_many_naive_pivot<double,432><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 432;
    break;
  case 28:
    inverse_many_naive_pivot<double,448><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 448;
    break;
  case 29:
    inverse_many_naive_pivot<double,464><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 464;
    break;
  case 30:
    inverse_many_naive_pivot<double,480><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 480;
    break;
  case 31:
    inverse_many_naive_pivot<double,496><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 496;
    break;
  case 32:
    inverse_many_naive_pivot<double,512><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 512;
    break;
  case 33:
    inverse_many_naive_pivot<double,528><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 528;
    break;
  case 34:
    inverse_many_naive_pivot<double,544><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 544;
    break;
  case 35:
    inverse_many_naive_pivot<double,560><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 560;
    break;
  case 36:
    inverse_many_naive_pivot<double,576><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 576;
    break;
  case 37:
    inverse_many_naive_pivot<double,592><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 592;
    break;
  case 38:
    inverse_many_naive_pivot<double,608><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 608;
    break;
  case 39:
    inverse_many_naive_pivot<double,624><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 624;
    break;
  case 40:
    inverse_many_naive_pivot<double,640><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 640;
    break;
  case 41:
    inverse_many_naive_pivot<double,656><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 656;
    break;
  case 42:
    inverse_many_naive_pivot<double,672><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 672;
    break;
  case 43:
    inverse_many_naive_pivot<double,688><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 688;
    break;
  case 44:
    inverse_many_naive_pivot<double,704><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 704;
    break;
  case 45:
    inverse_many_naive_pivot<double,720><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 720;
    break;
  case 46:
    inverse_many_naive_pivot<double,736><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 736;
    break;
    // case 47:
    //   inverse_many_naive_pivot<double,752><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 752;
    //   break;
    // case 48:
    //   inverse_many_naive_pivot<double,768><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 768;
    //   break;
    // case 49:
    //   inverse_many_naive_pivot<double,784><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 784;
    //   break;
    // case 50:
    //   inverse_many_naive_pivot<double,800><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 800;
    //   break;
  default:
    fprintf (stderr, "N=%d is larger than maximum 736 in cuda_inverse_many_double.\n");
  };
#ifdef CHECK_INVERSES
  // Check inverses for correctness
  // Copy A matrix pointers to worklist_d
  cudaMemcpyAsync (worklist_d, Alist_h, num_mats*sizeof(double*),
                   cudaMemcpyHostToDevice);
  // Call kernel to check (A^(-1) * A - I).  It returns 0 or 1 in
  // the Alist_d pointer.
  dim3 checkBlock(32);
  dim3 checkGrid(num_mats);
  check_inv<double,float,32><<<checkGrid,checkBlock>>>((double**)Alist_d, worklist_d, N, N_double, row_stride);
  cudaMemcpy (bad_inverse, Alist_d, num_mats*sizeof(double*),
              cudaMemcpyDeviceToHost);
  for (int mat=0; mat<num_mats; mat++)
  {
    if (bad_inverse[mat])
    {
      char name[1000];
      gethostname(name, 1000);
      fprintf (stderr, "Offending hostname = %s\n", name);
      std::cerr << "bad inverse for matrix " << mat << std::endl;
      std::vector<float>  Amat(N*row_stride);
      cudaMemcpy (&(Amat[0]), Alist_h[mat], N*row_stride*sizeof(float), cudaMemcpyDeviceToHost);
      std::ostringstream matName;
      matName << "BadMat_" << mat << ".dat";
      FILE *fout = fopen (matName.str().c_str(), "w");
      for (int row=0; row<N; row++)
      {
        for (int col=0; col<N; col++)
        {
          //fprintf (stderr, "row=%d col=%d\n", row, col);
          fprintf (fout, "%24.16e ", Amat[row*row_stride+col]);
        }
        fprintf (fout, "\n");
      }
      fclose (fout);
      std::vector<double> Ainv (N*N_double);
      cudaMemcpy (&(Ainv[0]), worklist_h[mat], N*N_double*sizeof(double), cudaMemcpyDeviceToHost);
      std::ostringstream invName;
      invName << "BadInv_" << mat << ".dat";
      fout = fopen (invName.str().c_str(), "w");
      for (int row=0; row<N; row++)
      {
        for (int col=0; col<N; col++)
        {
          //fprintf (stderr, "row=%d col=%d\n", row, col);
          fprintf (fout, "%24.16e ", Ainv[row*N_double+col]);
        }
        fprintf (fout, "\n");
      }
      fclose (fout);
    }
  }
#endif
  // Copy original pointer lists back to device
  hipMemcpyAsync (Alist_d, Alist_h, num_mats*sizeof(float*),
                   hipMemcpyHostToDevice);
  hipMemcpyAsync (worklist_d, worklist_h, num_mats*sizeof(float*),
                   hipMemcpyHostToDevice);
  dim3 dimGridConvert2((N*row_stride+(CONVERT_BS-1))/CONVERT_BS, num_mats);
  // Convert back to single precision.
  convert<<<dimGridConvert2,dimBlockConvert>>>
  (Alist_d, (double**) worklist_d,
   N, N, row_stride,
   N_double, N_double, N_double);
}




void
cuda_inverse_many_complex_double (float *Alist_d[], float *worklist_d[],
                                  int N, int row_stride, int num_mats)
{
  int N_double = ((N + INVERSE_BS-1)/INVERSE_BS)*INVERSE_BS;
  dim3 dimBlockConvert (CONVERT_BS);
  dim3 dimGridConvert((N_double*N_double+(CONVERT_BS-1))/CONVERT_BS,
                      num_mats);
  convert<<<dimGridConvert,dimBlockConvert>>>
  ((double**)worklist_d, Alist_d,
   N_double, N_double, N_double,
   N, N, row_stride);
  // We need to replace the pointers to the single-precision A
  // with the double-precision version we just converted.  We
  // Also need generate a new set of workspace pointers to point
  // to the region after the double-precision A.
  float *Alist_new[num_mats], *Alist_h[num_mats];
  float *worklist_h[num_mats];
  double *worklist_double_h[num_mats];
  float *bad_inverse[num_mats];
  // Save the original pointer lists on the host
  hipMemcpy (worklist_h, worklist_d, num_mats*sizeof(float*),
              hipMemcpyDeviceToHost);
  hipMemcpy (Alist_h, Alist_d, num_mats*sizeof(float*),
              hipMemcpyDeviceToHost);
  // Create new pointers as discussed above
  for (int i=0; i<num_mats; i++)
  {
    Alist_new[i] = worklist_h[i];
    worklist_double_h[i] = (double*)(worklist_h[i]) +N_double*N_double;
  }
  hipMemcpyAsync (worklist_d, worklist_double_h, num_mats*sizeof(double*),
                   hipMemcpyHostToDevice);
  hipMemcpyAsync (Alist_d, Alist_new, num_mats*sizeof(double*),
                   hipMemcpyHostToDevice);
  // Do the inversion in double-precision
  dim3 dimGrid(num_mats);
  int NB = (N+15)/16;
  int BS=0;
  dim3 dimBlock(NB*16);
  switch (NB)
  {
  case 1:
    complex_inverse_many_naive_pivot<double,16><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=16;
    break;
  case 2:
    complex_inverse_many_naive_pivot<double,32><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=32;
    break;
  case 3:
    complex_inverse_many_naive_pivot<double,48><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=48;
    break;
  case 4:
    complex_inverse_many_naive_pivot<double,64><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=64;
    break;
  case 5:
    complex_inverse_many_naive_pivot<double,80><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=80;
    break;
  case 6:
    complex_inverse_many_naive_pivot<double,96><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=96;
    break;
  case 7:
    complex_inverse_many_naive_pivot<double,112><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=112;
    break;
  case 8:
    complex_inverse_many_naive_pivot<double,128><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=128;
    break;
  case 9:
    complex_inverse_many_naive_pivot<double,144><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=144;
    break;
  case 10:
    complex_inverse_many_naive_pivot<double,160><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=160;
    break;
  case 11:
    complex_inverse_many_naive_pivot<double,176><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=176;
    break;
  case 12:
    complex_inverse_many_naive_pivot<double,192><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=192;
    break;
  case 13:
    complex_inverse_many_naive_pivot<double,208><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 208;
    break;
  case 14:
    complex_inverse_many_naive_pivot<double,224><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=224;
    break;
  case 15:
    complex_inverse_many_naive_pivot<double,240><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=240;
    break;
  case 16:
    complex_inverse_many_naive_pivot<double,256><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=256;
    break;
  case 17:
    complex_inverse_many_naive_pivot<double,272><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=272;
    break;
  case 18:
    complex_inverse_many_naive_pivot<double,288><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=288;
    break;
  case 19:
    complex_inverse_many_naive_pivot<double,304><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=304;
    break;
  case 20:
    complex_inverse_many_naive_pivot<double,320><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=320;
    break;
  case 21:
    complex_inverse_many_naive_pivot<double,336><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS=336;
    break;
  case 22:
    complex_inverse_many_naive_pivot<double,352><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 352;
    break;
  case 23:
    complex_inverse_many_naive_pivot<double,368><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 368;
    break;
  case 24:
    complex_inverse_many_naive_pivot<double,384><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 384;
    break;
  case 25:
    complex_inverse_many_naive_pivot<double,400><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 400;
    break;
  case 26:
    complex_inverse_many_naive_pivot<double,416><<<dimGrid,dimBlock>>>
    ((double**)Alist_d, (double**)worklist_d, N, N_double);
    BS = 416;
    break;
    // case 27:
    //   complex_inverse_many_naive_pivot<double,432><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 432;
    //   break;
    // case 28:
    //   complex_inverse_many_naive_pivot<double,448><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 448;
    //   break;
    // case 29:
    //   complex_inverse_many_naive_pivot<double,464><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 464;
    //   break;
    // case 30:
    //   complex_inverse_many_naive_pivot<double,480><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 480;
    //   break;
    // case 31:
    //   complex_inverse_many_naive_pivot<double,496><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 496;
    //   break;
    // case 32:
    //   complex_inverse_many_naive_pivot<double,512><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 512;
    //   break;
    // case 33:
    //   complex_inverse_many_naive_pivot<double,528><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 528;
    //   break;
    // case 34:
    //   complex_inverse_many_naive_pivot<double,544><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 544;
    //   break;
    // case 35:
    //   complex_inverse_many_naive_pivot<double,560><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 560;
    //   break;
    // case 36:
    //   complex_inverse_many_naive_pivot<double,576><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 576;
    //   break;
    // case 37:
    //   complex_inverse_many_naive_pivot<double,592><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 592;
    //   break;
    // case 38:
    //   complex_inverse_many_naive_pivot<double,608><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 608;
    // break;
    // case 39:
    //   complex_inverse_many_naive_pivot<double,624><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 624;
    //   break;
    // case 40:
    //   complex_inverse_many_naive_pivot<double,640><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 640;
    //   break;
    // case 41:
    //   complex_inverse_many_naive_pivot<double,656><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 656;
    //   break;
    // case 42:
    //   complex_inverse_many_naive_pivot<double,672><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 672;
    //   break;
    // case 43:
    //   complex_inverse_many_naive_pivot<double,688><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 688;
    //   break;
    // case 44:
    //   complex_inverse_many_naive_pivot<double,704><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 704;
    //   break;
    // case 45:
    //   complex_inverse_many_naive_pivot<double,720><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 720;
    //   break;
    // case 46:
    //   complex_inverse_many_naive_pivot<double,736><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 736;
    //   break;
    // case 47:
    //   complex_inverse_many_naive_pivot<double,752><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 752;
    //   break;
    // case 48:
    //   complex_inverse_many_naive_pivot<double,768><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 768;
    //   break;
    // case 49:
    //   complex_inverse_many_naive_pivot<double,784><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 784;
    //   break;
    // case 50:
    //   complex_inverse_many_naive_pivot<double,800><<<dimGrid,dimBlock>>>
    // 	((double**)Alist_d, (double**)worklist_d, N, N_double);
    //   BS = 800;
    //   break;
  default:
    fprintf (stderr, "N=%d is larger than maximum 416 in cuda_complex_inverse_many_double.\n");
  };
#ifdef CHECK_INVERSES
  // Check inverses for correctness
  // Copy A matrix pointers to worklist_d
  cudaMemcpyAsync (worklist_d, Alist_h, num_mats*sizeof(double*),
                   cudaMemcpyHostToDevice);
  // Call kernel to check (A^(-1) * A - I).  It returns 0 or 1 in
  // the Alist_d pointer.
  dim3 checkBlock(32);
  dim3 checkGrid(num_mats);
  check_inv<double,float,32><<<checkGrid,checkBlock>>>((double**)Alist_d, worklist_d, N, N_double, row_stride);
  cudaMemcpy (bad_inverse, Alist_d, num_mats*sizeof(double*),
              cudaMemcpyDeviceToHost);
  for (int mat=0; mat<num_mats; mat++)
    if (bad_inverse[mat])
    {
      char name[1000];
      gethostname(name, 1000);
      fprintf (stderr, "Offending hostname = %s\n", name);
      std::cerr << "bad inverse for matrix " << mat << std::endl;
      std::vector<float>  Amat(N*row_stride);
      cudaMemcpy (&(Amat[0]), Alist_h[mat], N*row_stride*sizeof(float), cudaMemcpyDeviceToHost);
      std::ostringstream matName;
      matName << "BadMat_" << mat << ".dat";
      FILE *fout = fopen (matName.str().c_str(), "w");
      for (int row=0; row<N; row++)
      {
        for (int col=0; col<N; col++)
        {
          //fprintf (stderr, "row=%d col=%d\n", row, col);
          fprintf (fout, "%24.16e ", Amat[row*row_stride+col]);
        }
        fprintf (fout, "\n");
      }
      fclose (fout);
      std::vector<double> Ainv (N*N_double);
      cudaMemcpy (&(Ainv[0]), worklist_h[mat], N*N_double*sizeof(double), cudaMemcpyDeviceToHost);
      std::ostringstream invName;
      invName << "BadInv_" << mat << ".dat";
      fout = fopen (invName.str().c_str(), "w");
      for (int row=0; row<N; row++)
      {
        for (int col=0; col<N; col++)
        {
          //fprintf (stderr, "row=%d col=%d\n", row, col);
          fprintf (fout, "%24.16e ", Ainv[row*N_double+col]);
        }
        fprintf (fout, "\n");
      }
      fclose (fout);
    }
#endif
  // Copy original pointer lists back to device
  hipMemcpyAsync (Alist_d, Alist_h, num_mats*sizeof(float*),
                   hipMemcpyHostToDevice);
  hipMemcpyAsync (worklist_d, worklist_h, num_mats*sizeof(float*),
                   hipMemcpyHostToDevice);
  dim3 dimGridConvert2((N*row_stride+(CONVERT_BS-1))/CONVERT_BS, num_mats);
  // Convert back to single precision.
  convert<<<dimGridConvert2,dimBlockConvert>>>
  (Alist_d, (double**) worklist_d,
   N, N, row_stride,
   N_double, N_double, N_double);
}



void
cuda_inverse_many_double (double *Alist_d[], double *worklist_d[],
                          int N, int num_mats)
{
  dim3 dimBlock(INVERSE_BS,2);
  dim3 dimGrid(num_mats);
  inverse_many_pivot<double,INVERSE_BS><<<dimGrid,dimBlock>>>
  (Alist_d, worklist_d, N, N);
}




void
cuda_inverse_many (double *Alist_d[], double *worklist_d[],
                   int N, int num_mats)
{
  dim3 dimBlock(INVERSE_BS);
  dim3 dimGrid(num_mats);
  inverse_many_pivot<double,INVERSE_BS><<<dimGrid,dimBlock>>>
  (Alist_d, worklist_d, N, N);
}



//////////////////////////////////////////////////////
//                  Test routines                   //
//////////////////////////////////////////////////////



#ifdef CUDA_TEST_MAIN

void
test_inverse()
{
  int N = 32;
  dim3 dimBlock(INVERSE_BS,2);
  dim3 dimGrid(1);
  float *A_d, *work_d;
  int lwork = N*N + INVERSE_BS * INVERSE_BS;
  cudaMalloc((void**)&A_d, N*N*sizeof(float));
  cudaMalloc((void**)&work_d, lwork*sizeof(float));
  float A[N*N], Ainv[N*N];
  for (int i=0; i<N*N; i++)
    A[i] = drand48();
  cudaMemcpyAsync(A_d, A, N*N*sizeof(float), cudaMemcpyHostToDevice);
  inverse<float,INVERSE_BS><<<dimGrid,dimBlock>>> (A_d, work_d, N, N);
  // Copy Ainv back to host memory
  cudaMemcpy(Ainv, A_d, N*N*sizeof(float), cudaMemcpyDeviceToHost);
  float error = 0.0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
    {
      float val = 0.0;
      for (int k=0; k<N; k++)
        val += Ainv[i*N+k]*A[k*N+j];
      float diff = (i==j) ? (1.0f-val) : val;
      error += diff*diff;
    }
  fprintf (stderr, "error = %1.8e\n", sqrt(error/(double)(N*N)));
}



void
test_inverse_many()
{
  int numMats = 10000;
  int N = 128;
  int lwork = N*N + INVERSE_BS * INVERSE_BS;
  fprintf (stderr, "lwork = %d\n", lwork);
  float **Alist, **worklist;
  float **Alist_d, **worklist_d;
  Alist    = (float**)malloc(numMats*sizeof(float*));
  worklist = (float**)malloc(numMats*sizeof(float*));
  cudaMalloc((void**)&Alist_d,    numMats*sizeof(float*));
  cudaMalloc((void**)&worklist_d, numMats*sizeof(float*));
  float A[N*N];
  for (int i=0; i<N*N; i++)
    A[i] = drand48();
  for (int mat=0; mat<numMats; mat++)
  {
    cudaMalloc ((void**)&(Alist[mat]),    N*N*sizeof(float));
    cudaMalloc ((void**)&(worklist[mat]), lwork*sizeof(float));
    cudaMemcpyAsync(Alist[mat], A, N*N*sizeof(float), cudaMemcpyHostToDevice);
  }
  cudaMemcpyAsync(Alist_d   ,    Alist, numMats*sizeof(float*),
                  cudaMemcpyHostToDevice);
  cudaMemcpyAsync(worklist_d, worklist, numMats*sizeof(float*),
                  cudaMemcpyHostToDevice);
  dim3 dimBlock(INVERSE_BS,2);
  dim3 dimGrid(numMats);
  clock_t start = clock();
  for (int i=0; i<1; i++)
  {
    inverse_many_pivot<float,INVERSE_BS><<<dimGrid,dimBlock>>>
    (Alist_d, worklist_d, N, N);
//     inverse_many<float,INVERSE_BS><<<dimGrid,dimBlock>>>
//       (Alist_d, worklist_d, N, N);
  }
  cudaThreadSynchronize();
  clock_t end = clock();
  double time = (double)(end-start)/(double)CLOCKS_PER_SEC
                / (double)numMats;
  double rate = 1.0/time;
  fprintf (stderr, "Rate is %1.3f matrix inversions per second.\n",
           rate);
  cudaError_t err = cudaGetLastError();
  if (err != cudaSuccess)
  {
    fprintf (stderr, "CUDA error in block_inverse:\n  %s\n",
             cudaGetErrorString(err));
    abort();
  }
  // Copy Ainv back to host memory
  float Ainv[N*N];
  cudaMemcpy(Ainv, Alist[10], N*N*sizeof(float), cudaMemcpyDeviceToHost);
  float error = 0.0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
    {
      float val = 0.0;
      for (int k=0; k<N; k++)
        val += Ainv[i*N+k]*A[k*N+j];
      float diff = (i==j) ? (1.0f-val) : val;
      error += diff*diff;
    }
  fprintf (stderr, "error = %1.8e\n", sqrt(error/(double)(N*N)));
}


void
test_inverse_many_double()
{
  int numMats = 1000;
  int N = 128;
  int lwork = N*N + INVERSE_BS * INVERSE_BS;
  fprintf (stderr, "lwork = %d\n", lwork);
  double **Alist, **worklist;
  double **Alist_d, **worklist_d;
  Alist    = (double**)malloc(numMats*sizeof(double*));
  worklist = (double**)malloc(numMats*sizeof(double*));
  cudaMalloc((void**)&Alist_d,    numMats*sizeof(double*));
  cudaMalloc((void**)&worklist_d, numMats*sizeof(double*));
  double A[N*N];
  for (int i=0; i<N*N; i++)
    A[i] = drand48();
  for (int mat=0; mat<numMats; mat++)
  {
    cudaMalloc ((void**)&(Alist[mat]),    N*N*sizeof(double));
    cudaMalloc ((void**)&(worklist[mat]), lwork*sizeof(double));
    cudaMemcpyAsync(Alist[mat], A, N*N*sizeof(double), cudaMemcpyHostToDevice);
  }
  cudaMemcpyAsync(Alist_d   ,    Alist, numMats*sizeof(double*),
                  cudaMemcpyHostToDevice);
  cudaMemcpyAsync(worklist_d, worklist, numMats*sizeof(double*),
                  cudaMemcpyHostToDevice);
  dim3 dimBlock(INVERSE_BS,2);
  dim3 dimGrid(numMats);
  clock_t start = clock();
  for (int i=0; i<1; i++)
  {
    inverse_many_pivot<double,INVERSE_BS><<<dimGrid,dimBlock>>>
    (Alist_d, worklist_d, N, N);
//     inverse_many<double,INVERSE_BS><<<dimGrid,dimBlock>>>
//       (Alist_d, worklist_d, N, N);
  }
  cudaThreadSynchronize();
  clock_t end = clock();
  double time = (double)(end-start)/(double)CLOCKS_PER_SEC
                / (double)numMats;
  double rate = 1.0/time;
  fprintf (stderr, "Rate is %1.3f matrix inversions per second.\n",
           rate);
  cudaError_t err = cudaGetLastError();
  if (err != cudaSuccess)
  {
    fprintf (stderr, "CUDA error in block_inverse:\n  %s\n",
             cudaGetErrorString(err));
    abort();
  }
  // Copy Ainv back to host memory
  double Ainv[N*N];
  cudaMemcpy(Ainv, Alist[10], N*N*sizeof(double), cudaMemcpyDeviceToHost);
  double error = 0.0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
    {
      double val = 0.0;
      for (int k=0; k<N; k++)
        val += Ainv[i*N+k]*A[k*N+j];
      double diff = (i==j) ? (1.0f-val) : val;
      error += diff*diff;
    }
  fprintf (stderr, "error = %1.8e\n", sqrt(error/(double)(N*N)));
}




void
test_inverse_many_double_conv()
{
  srand48((long) 12394);
  int numMats = 100;
  int N = 244;
  int row_stride = 256;
  int lwork = cuda_inverse_many_double_worksize(N);
  fprintf (stderr, "lwork = %d\n", lwork);
  float **Alist, **worklist;
  float **Alist_d, **worklist_d;
  Alist    = (float**)malloc(numMats*sizeof(float*));
  worklist = (float**)malloc(numMats*sizeof(float*));
  cudaMalloc((void**)&Alist_d,    numMats*sizeof(float*));
  cudaMalloc((void**)&worklist_d, numMats*sizeof(float*));
  float *A = (float*)malloc(sizeof(float)*numMats*N*row_stride);
  for (int j=0; j<numMats; j++)
    for (int i=0; i<N*row_stride; i++)
      A[j*N*row_stride+i] = 1.0*(drand48()-0.5);
  for (int mat=0; mat<numMats; mat++)
  {
    cudaMalloc ((void**)&(Alist[mat]),    N*row_stride*sizeof(float));
    cudaMalloc ((void**)&(worklist[mat]), lwork*sizeof(float));
    cudaMemcpyAsync(Alist[mat], &A[mat*N*row_stride], N*row_stride*sizeof(float), cudaMemcpyHostToDevice);
  }
  cudaMemcpyAsync(Alist_d   ,    Alist, numMats*sizeof(float*),
                  cudaMemcpyHostToDevice);
  cudaMemcpyAsync(worklist_d, worklist, numMats*sizeof(float*),
                  cudaMemcpyHostToDevice);
  dim3 dimBlock(INVERSE_BS,2);
  dim3 dimGrid(numMats);
  clock_t start = clock();
  for (int i=0; i<1; i++)
  {
    cuda_inverse_many_double (Alist_d, worklist_d, N, row_stride, numMats);
    //    cuda_inverse_many_double (Alist_d, worklist_d, N, numMats);
  }
  cudaThreadSynchronize();
  clock_t end = clock();
  double t = (double)(end-start)/(double)CLOCKS_PER_SEC / (double)numMats;
  double rate = 1.0/t;
  fprintf (stderr, "Rate is %1.3f matrix inversions per second.\n",
           rate);
  cudaError_t err = cudaGetLastError();
  if (err != cudaSuccess)
  {
    fprintf (stderr, "CUDA error in block_inverse:\n  %s\n",
             cudaGetErrorString(err));
    abort();
  }
  // Copy Ainv back to host memory
  for (int mat=0; mat<numMats; mat++)
  {
    float Ainv[N*row_stride];
    cudaMemcpy(Ainv, Alist[mat], N*row_stride*sizeof(float), cudaMemcpyDeviceToHost);
    double error = 0.0;
    for (int i=0; i<N; i++)
      for (int j=0; j<N; j++)
      {
        double val = 0.0;
        for (int k=0; k<N; k++)
          val += Ainv[i*row_stride+k]*A[mat*N*row_stride+k*row_stride+j];
        double diff = (i==j) ? (1.0f-val) : val;
        error += diff*diff;
      }
    fprintf (stderr, "error = %1.8e\n", sqrt(error/(double)(N*N)));
  }
}






#include <stdio.h>

main()
{
  //test_inverse_many();
  test_inverse_many_double_conv();
  //test_inverse_many_double();
  // int N=32;
  // float A[N*N], Acopy[N*N];
  // float *A_d;
  // for (int i=0; i<N; i++)
  //   for (int j=0; j<N; j++)
  //     A[N*i+j] = Acopy[N*i+j] = (float) drand48();
  // cudaMalloc ((void**)&A_d, N*N*sizeof(float));
  // cudaMemcpyAsync (A_d, A, N*N*sizeof(float),
  // 	      cudaMemcpyHostToDevice);
  // dim3 dimBlock(N);
  // dim3 dimGrid(1);
  // block_inverse<float,32><<<dimGrid,dimBlock>>> (A_d, N, N);
  // cudaThreadSynchronize();
  // cudaError_t err = cudaGetLastError();
  // if (err != cudaSuccess) {
  //   fprintf (stderr, "CUDA error in block_inverse:\n  %s\n",
  // 	     cudaGetErrorString(err));
  //   abort();
  // }
  // cudaMemcpy (A, A_d, N*N*sizeof(float),
  // 	      cudaMemcpyDeviceToHost);
  // float nrm = 0.0;
  // for (int i=0; i<N; i++)
  //   for (int j=0; j<N; j++) {
  //     float val = 0.0;
  //     for (int k=0; k<N; k++)
  // 	val += A[i*N+k] * Acopy[k*N+j];
  //     float diff = (i==j) ? 1.0-val : val;
  //     nrm += diff*diff;
  //   }
  // fprintf (stderr, "Error = %1.6e\n", sqrt(nrm/(double)(N*N)));
}
#endif
